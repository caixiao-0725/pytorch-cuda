#include "hip/hip_runtime.h"
#include "include/utils.h"

template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n>=feats.size(0) || f>=feats.size(2)) return;

    // point -1~1
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;
    
    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;
    feat_interp[n][f] = (1-u)*(a*feats[n][0][f] +
                               b*feats[n][1][f] +
                               c*feats[n][2][f] +
                               d*feats[n][3][f]) + 
                            u*(a*feats[n][4][f] +
                               b*feats[n][5][f] +
                               c*feats[n][6][f] +
                               d*feats[n][7][f]);
}


torch::Tensor trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor point) {

    //第0维
    const int N = feats.size(0);
    const int F = feats.size(2);

    torch::Tensor feat_interp = torch::empty({N, F}, feats.options());
    //torch::zeros({N, F},torch::dtype(torch::kInt32).device(feats.device()));
    const dim3 threads(16,16); //256
    //const dim3 threads(256); //256
    const dim3 blocks((N + threads.x - 1) / threads.x, (F + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cuda", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            feats.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
            point.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            feat_interp.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>()
            
        //trilinear_fw_kernel<<<blocks, threads>>>(
        //    feats.packed_accessor<float,3,torch::RestrictPtrTraits>(),
        //    point.packed_accessor<float,2,torch::RestrictPtrTraits>(),
        //    feat_interp.packed_accessor<float,2,torch::RestrictPtrTraits>());
        );
    }));
    return feat_interp;
}

